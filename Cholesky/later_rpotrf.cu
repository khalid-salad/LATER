#include "hip/hip_runtime.h"
#include "LATER.h"
#include <assert.h>
#include <hip/hip_fp16.h>

#define BLOCKSIZE 2048
#define LWORK 65536

int chol_info;
int lwork;

int *dev_info;

float chol_panel = 0.0;
float chol_gemm = 0.0;

/*
This function performs recursive Cholesky factorization
*/

void u_potrf(cudaCtxt ctxt, int n, float *A, int lda, float *work,
             __half *hwork) {
  float ms;
  if (n <= BLOCKSIZE) {
    hipsolverDnSpotrf(ctxt.cusolver_handle, HIPBLAS_FILL_MODE_LOWER, n, A, lda,
                     work, LWORK, dev_info);

    return;
  } else {
    auto n1 = n / 2;
    auto n2 = n - n1;
    u_potrf(ctxt, n1, A, lda, work, hwork);
    auto A12 = A + n * n1;
    auto A22 = A + n1 + n * n1;
    later_rtrsm(ctxt.cublas_handle, 'l', 'l', 't', n1, n2, A, lda, A12, lda,
                hwork);
    later_rsyrk(ctxt.cublas_handle, n2, n1, -1.0, A12, lda, 1.0, A22, lda,
                hwork);
    ms = stopTimer();
    chol_gemm += ms;
    auto tflops = 2.0 * n / 2 * n / 2 * n / 2 / ms / 1e9;
    u_potrf(ctxt, n2, A22, lda, work, hwork);
  }
}

void l_potrf(cudaCtxt ctxt, int n, float *A, int lda, float *work,
             __half *hwork) {
  float ms;
  if (n <= BLOCKSIZE) {
    startTimer();
    hipsolverDnSpotrf(ctxt.cusolver_handle, HIPBLAS_FILL_MODE_LOWER, n, A, lda,
                     work, LWORK, dev_info);

    chol_panel += stopTimer();

    return;
  } else {
    auto n1 = n / 2;
    auto n2 = n - n1;
    l_potrf(ctxt, n1, A, lda, work, hwork);
    startTimer();
    auto A21 = A + n1;
    auto A22 = A21 + n1 + n * n1;
    later_rtrsm(ctxt.cublas_handle, 'l', 'r', 't', n2, n1, A, lda, A21, lda,
                hwork);
    later_rsyrk(ctxt.cublas_handle, n2, n1, -1.0, A21, lda, 1.0, A22, lda,
                hwork);
    ms = stopTimer();
    chol_gemm += ms;
    auto tflops = 2.0 * n / 2 * n / 2 * n / 2 / ms / 1e9;
    l_potrf(ctxt, n2, A22, lda, work, hwork);
  }
}

void later_rpotrf(char uplo, int n, float *A, int lda, float *work,
                  __half *hwork) {
  cudaCtxt ctxt;
  hipblasCreate(&ctxt.cublas_handle);
  hipsolverDnCreate(&ctxt.cusolver_handle);
  // printMatrixDeviceBlock("A.csv", n,n, A,n);
  hipMalloc(&dev_info, sizeof(int));

  if (uplo == 'l') {
    l_potrf(ctxt, n, A, lda, work, hwork);
  } else if (uplo == 'u') {
    u_potrf(ctxt, n, A, lda, work, hwork);
  }

  hipblasDestroy(ctxt.cublas_handle);
  hipsolverDnDestroy(ctxt.cusolver_handle);
  return;
}