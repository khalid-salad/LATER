#include "hip/hip_runtime.h"
#include "LATER.h"

int n;
bool checkFlag = false;

int parseArguments(int argc, char *argv[]) {
  n = atoi(argv[1]);
  printf("n = %d\n", n);
  for (int i = 2; i < argc; i++) {
    if (strcmp(argv[i], "-check") == 0) {
      checkFlag = true;
    }
  }
  return 0;
}

int main(int argc, char *argv[]) {
  if (argc < 2) {
    printf("Usage: ./test_potrf n [options]\n");
    printf("Options:\n\t-check: enable checking the orthogonality and backward "
           "error\n");
    return EXIT_FAILURE;
  }
  if (parseArguments(argc, argv) != 0) {
    return EXIT_FAILURE;
  }
  print_env();
  float *A;
  hipMalloc(&A, sizeof(*A) * n * n);
  slowGenerateUniformPositiveDefiniteMatrix(A, n);

  float *twork;
  hipMalloc(&twork, sizeof(float) * n * n);

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  float sone = 1.0;
  float snegone = -1.0;
  float szero = 0.0;

  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, n, n, &sone, A, n, A, n,
              &szero, twork, n);

  hipMemcpy(A, twork, sizeof(float) * n * n, hipMemcpyDeviceToDevice);

  float normA = snorm(n, n, A);

  float *work;
  hipMalloc(&work, sizeof(*work) * 128 * 128);

  __half *hwork;
  hipMalloc(&hwork, sizeof(*hwork) * n / 2 * n);

  // printMatrixDeviceBlock("AA.csv", n, n, A, n);

  printf("n = %d\n", n);
  startTimer();
  later_rpotrf('l', n, A, n, work, hwork);
  auto ms = stopTimer();
  // printMatrixDeviceBlock("LL.csv", n, n, A, n);

  if (checkFlag) {
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, n, n, &snegone, A, n, A, n,
                &sone, twork, n);
    printf("Backward error ||LL^T-A||/||A|| = %.6e\n",
           snorm(n, n, twork) / normA);
  }

  hipFree(A);
  hipFree(twork);
  hipFree(work);
  hipFree(hwork);
}